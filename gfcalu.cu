#include "hip/hip_runtime.h"
#include "gfcalu.cuh"
#include <stdio.h>

__device__
const int mgfaddtable[4][4] = {{0,1,2,3},{1,0,3,2},{2,3,0,1},{3,2,1,0}};
__device__
const int mgfsubtable[4][4] = {{0,1,2,3},{1,0,3,2},{2,3,0,1},{3,2,1,0}};
__device__
const int mgfmultable[4][4] = {{0,0,0,0},{0,1,2,3},{0,2,3,1},{0,3,1,2}};
__device__
const int mgfdivtable[4][4] = {{0,0,0,0},{0,1,3,2},{0,2,1,3},{0,3,2,1}};

__device__
int mgfadd(int a, int b)
{
	return mgfaddtable[a][b];
}
__device__
int mgfsub(int a, int b)
{
	return mgfsubtable[a][b];
}
__device__
int mgfmul(int a, int b)
{
	return mgfmultable[a][b];
}
__device__
int mgfdiv(int a, int b)
{
	return mgfdivtable[a][b];
}

__global__
void gfmatrixmulKernel(int *d_c,int *d_H, int *d_flag, int *d_m2n, int N, int M)
/*H和c展开成一维向量
矩阵是N行M列*/
{
    int tid = threadIdx.x + blockIdx.x* blockDim.x; //每个线程计算一行的结果
    int sum = 0;
    if(tid < N)
    {
        for (int i = 0; i < 5; i ++)
        {
            sum = mgfadd(sum,mgfmul(d_c[d_m2n[tid*5 + i]-1],d_H[tid*M + d_m2n[tid*5 + i]-1]));
        }
        d_flag[tid] = sum;
    }
}