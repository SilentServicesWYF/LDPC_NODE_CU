#include "aux.h"
#include "gfcalu.cuh"
#include "llrv.cuh"
#include <iostream>
#include <algorithm>
#include <thrust/device_vector.h>
#include <time.h>
#include <sys/time.h>

//参数设定
int row1 = 1344;
int col1 = 2688;
int row2 = 2688;
int col2 = 1;
const int maxweight1 = 3;
const int maxweight2 = 5;
const int pskdict[8] = {-1,-1,-1,1,1,-1,1,1};

int main()
{
	// 初始化内存
	int *H = new int [row1*col1];
	int *c = new int [col1];
	int *m2n = new int [row1*maxweight2];
	int *n2m = new int [row2*maxweight1];
	float *constell = new float [col1*2];
	int *n2m_num = new int [col1];
	int *m2n_num = new int [row1];

	readvector(col1,n2m_num,"data/n2m_num.txt");
	readvector(row1,m2n_num,"data/m2n_num.txt");
	readvector(row1*col1,H,"data/H.txt");
	readvector(col1,c,"data/c.txt");
	readvector(row1*maxweight2,m2n,"data/m2n.txt");
	readvector(row2*maxweight1,n2m,"data/n2m.txt");
	readconstell(col1*2,constell,"data/constell.txt");

	/*复制m2n,n2m,m2n_num,n2m_num,H到显存*/
	int *d_m2n = 0;
	int *d_n2m = 0;
	int *d_m2n_num = 0;
	int *d_n2m_num = 0;
	int *d_H = 0;

	hipMalloc(&d_m2n, sizeof(int)*row1*maxweight2);
	hipMalloc(&d_n2m, sizeof(int)*row2*maxweight1);
	hipMalloc(&d_m2n_num, sizeof(int)*row1);
	hipMalloc(&d_n2m_num, sizeof(int)*col1);
	hipMalloc(&d_H, sizeof(int)*row1*col1);

	hipMemcpy(d_m2n, m2n, sizeof(int)*row1*maxweight2, hipMemcpyHostToDevice);
	hipMemcpy(d_n2m, n2m, sizeof(int)*row2*maxweight1, hipMemcpyHostToDevice);
	hipMemcpy(d_m2n_num, m2n_num, sizeof(int)*row1, hipMemcpyHostToDevice);
	hipMemcpy(d_n2m_num, n2m_num, sizeof(int)*col1, hipMemcpyHostToDevice);
	hipMemcpy(d_H, H, sizeof(int)*col1*row1, hipMemcpyHostToDevice);

	//主机端计算Lch
	float *subconstell;
	float *Lch = new float [col1*4];
	for (int n = 0; n < col1; n++)
	{
		subconstell =  floatslice(constell, n*2, n*2+1);
		for (int i = 0; i < 4; i ++)
		{
			Lch[n*4+i] = LLRV(subconstell, pskdict, i);
		}
		delete []subconstell;
	}
	//把Lch放到显存中
	float *d_Lch = 0;
	hipMalloc(&d_Lch, col1*4*sizeof(float));
	hipMemcpy(d_Lch, Lch, col1*4*sizeof(float), hipMemcpyHostToDevice);

	/*在主机端初始化Ln2m,Lm2n,Ln2mbuff*/
	float *Ln2m = new float [row2*maxweight1*4]();
	float *Lm2n = new float [row1*maxweight2*4]();
	float *Ln2mbuff = new float [row2*maxweight1*4]();
	for (int k = 0; k < row2; k ++)
	{
		int avm_index = 0;
		while ((n2m[k*maxweight1+avm_index] != 0) && (avm_index < maxweight1))
		{
			int Lch_index = 0;
			for (int buff_index = avm_index*4; buff_index < (avm_index + 1)*4; buff_index ++)
			{
				Ln2mbuff[k*maxweight1*4+buff_index] = Lch[k*4+Lch_index];
				Lch_index = Lch_index + 1;
			}
			avm_index = avm_index + 1;
		}
	}
	/*复制Ln2m,Lm2n,Ln2mbuff到显存*/
	float *d_Ln2m = 0;
	float *d_Lm2n = 0;
	float *d_Ln2mbuff = 0;
	hipMalloc(&d_Ln2m, sizeof(float)*row2*maxweight1*4);
	hipMalloc(&d_Lm2n, sizeof(float)*row1*maxweight2*4);
	hipMalloc(&d_Ln2mbuff, sizeof(float)*row2*maxweight1*4);
	hipMemcpy(d_Ln2m, Ln2m, sizeof(float)*row2*maxweight1*4, hipMemcpyHostToDevice);
	hipMemcpy(d_Lm2n, Lm2n, sizeof(float)*row1*maxweight2*4, hipMemcpyHostToDevice);
	hipMemcpy(d_Ln2mbuff, Ln2mbuff, sizeof(float)*row2*maxweight1*4, hipMemcpyHostToDevice);

	//迭代参数
	int iterflag = 1;
	int iter_num = 0;
	int maxiter = 1;
	//为显存中的Lpost和est_c申请空间
	float *d_Lpost = 0; //每次赋值的时候直接从内存拷贝
	int *d_est_c = 0;
	int *h_est_c = new int[col1](); //矩阵相乘在设备端调用
	int *h_flag = new int [row1](); //矩阵相乘在设备端返回的结果
	int *d_flag = 0;
	int *d_c = 0;

	hipMalloc(&d_Lpost, col1*4*sizeof(float));
	hipMalloc(&d_est_c, col1*sizeof(int));
	hipMalloc(&d_flag, row1*sizeof(int));
	hipMalloc(&d_c, col1*sizeof(int));

	hipMemcpy(d_est_c, h_est_c, col1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_flag, h_flag ,row1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, col1*sizeof(int), hipMemcpyHostToDevice);

    while (iterflag == 1 && iter_num < maxiter)
    {
        iter_num = iter_num + 1;
        //为显存中的Lpost赋初值
		hipMemcpy(d_Lpost, d_Lch, col1*4*sizeof(float), hipMemcpyDeviceToDevice);
        // 更新Lpost
        int lblocksize = 32;
        int lgridsize = divup(col1, lblocksize);
        LpostupdateKernel<<<lgridsize, lblocksize>>>(d_Lpost, d_Lm2n, d_m2n, d_m2n_num, d_n2m, d_n2m_num);
        // dim3 blocksize(4,32);
        // dim3 gridsize(divup(4,4),divup(col1,32));
        // testKernel<<<gridsize,blocksize>>>(d_Lpost, col1, 4);
        // hipDeviceSynchronize();
        // 根据Lpost计算最大似然译码
		int cblocksize = 32;
		int cgridsize = divup(col1, cblocksize);
		estcKernel<<<cgridsize, cblocksize>>>(d_Lpost, d_est_c);
        // 水平信息传递
		// 初始化Ln2m
		hipMemcpy(d_Ln2m, d_Ln2mbuff, sizeof(float)*row2*maxweight1*4, hipMemcpyDeviceToDevice);
        //更新Ln2m
        int hblocksize = 32;
        int hgridsize = divup(col1, hblocksize);
        Ln2mupdateKernel<<<hblocksize, hgridsize>>>(d_Ln2m, d_Lm2n, d_n2m, d_m2n, d_n2m_num, d_m2n_num);
        // dim3 blocksize(12,32);
		// dim3 gridsize(divup(12,12),divup(row1,32));
		// testKernel<<<gridsize,blocksize>>>(d_Ln2m, col1, 12);
		// hipDeviceSynchronize();

        //垂直信息传递
        int vblocksize = 32;
        int vgridsize = divup(row1, vblocksize);
        Lm2nupdateKernel<<<vgridsize, vblocksize>>>(d_Lm2n, d_Ln2m, d_m2n, d_n2m, d_m2n_num, d_n2m_num, d_H);
        
    }
    return 0;
}