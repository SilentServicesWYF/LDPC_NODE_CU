#include "hip/hip_runtime.h"
#include "llrv.cuh"
#include "gfcalu.cuh"
#include <iostream>

//读显存专用函数
__global__
void testKernel(float *pointerArray, int row, int col)
{
	int c = threadIdx.x + blockIdx.x*blockDim.x;
	int r = threadIdx.y + blockIdx.y*blockDim.y;
	if (c < col && r < row)
	{
		printf("(%d,%d):%f\n",r+1,c+1,pointerArray[r*col+c]);
	}
}

__global__
void estcKernel(float *d_Lpost, int *d_est_c)
// 最大似然解码的函数,可以一维block 每block32个线程同时做完
{
	int n = blockIdx.x*blockDim.x + threadIdx.x;
	if (n < 2688)
	{
		float temp = d_Lpost[n*4];
		int index = 0;
		for (int k = 1; k < 4; k ++)
		{
			if (d_Lpost[n*4+k] > temp)
			{
				temp = d_Lpost[n*4+k];
				index = k;
			}
		}
		d_est_c[n] = index;
	}
}

__device__
void Lpostupdate(float *d_Lpost, float *d_Lm2n, int targetm2n, int n_index, int m_index)
{
    for (int k = 0; k < 4; k ++)
    {
        float targetLm2n = d_Lm2n[m_index*5*4 + targetm2n*4 + k];
        d_Lpost[n_index*4 + k] = d_Lpost[n_index*4 + k] + targetLm2n;
    }
}

__global__
void LpostupdateKernel(float *d_Lpost, float *d_Lm2n, int *d_m2n, int *d_m2n_num, int *d_n2m, int *d_n2m_num)
{
    int n_index = blockIdx.x*blockDim.x + threadIdx.x;
    if (n_index < 2688)
    {
        int mset_num = d_n2m_num[n_index];
        int mset[5];
        for (int k = 0; k < mset_num; k ++)
        {
            mset[k] = d_n2m[n_index*3 + k];
        }
        //搜索到所有与n连接的m节点之后对每一个被连接的m节点搜索其连接的n节点在Lm2n中的位置然后更新Lpost
        for (int k = 0; k < mset_num; k ++)
        {
            //搜索mset[k]对应的m2n的index
            int targetm2n = 0;
            for (int s = (mset[k]-1)*5; s < mset[k]*5; s ++)
            {
                if ((n_index + 1) == d_m2n[s])
                {
                    break;
                }
                targetm2n = targetm2n + 1;
            }
            Lpostupdate(d_Lpost, d_Lm2n, targetm2n, n_index, mset[k]-1);
        }
    }
}

__device__
void Ln2mupdate(float *d_Ln2m, float *d_Lm2n, int targetn2m, int targetm2n, int n_index, int m_index)
{
    for (int k = 0; k < 4; k ++)
    {
        float targetLm2n = d_Lm2n[m_index*5*4 + targetm2n*4 + k];
        d_Ln2m[n_index*3*4 + targetn2m*4 + k] = d_Ln2m[n_index*3*4 + targetn2m*4 + k] + targetLm2n;
    }
}

__global__
void Ln2mupdateKernel(float *d_Ln2m, float *d_Lm2n, int *d_n2m, int *d_m2n, int *d_n2m_num, int *d_m2n_num)
{
    int n_index = blockIdx.x*blockDim.x + threadIdx.x;
    if (n_index < 2688)
    {
        int mset_num = d_n2m_num[n_index];
        int mset[5];
        for (int k = 0; k < mset_num; k++)
        {
            mset[k] = d_n2m[n_index*3 + k];
        }
        //对每个n连接的m节点计算除了本m以外连接到n的m节点进行LLRV更新
        for (int k = 0; k < mset_num; k ++)
        {
            //搜索除了本m节点外的m节点
            int msubset[4];
            int msel = mset[k];
            int subset_index = 0;
            for (int kk = 0; kk < mset_num; kk ++)
            {
                if (msel != mset[kk])
                {
                    msubset[subset_index] = mset[kk];
                }
                subset_index = subset_index + 1;
            }
            for (int kk = 0; kk < mset_num - 1; kk ++)
            {
                //搜索msubset[kk]对应的m2n的index
                int targetm2n = 0;
                for (int s = (msubset[kk]-1)*5; s < msubset[kk]*5; s ++)
                {
                    if ((n_index + 1) == d_m2n[s])
                    {
                        break;
                    }
                    targetm2n = targetm2n + 1;
                }
                Ln2mupdate(d_Ln2m, d_Lm2n, k, targetm2n, n_index, msubset[kk]-1);
            }
        }
    }
}